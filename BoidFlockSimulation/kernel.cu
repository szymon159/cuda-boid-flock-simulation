#include "hip/hip_runtime.h"
#include "kernel.cuh"

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//void boidMoveKernelExecutor(float3 *&d_boids, size_t &arraySize, float dt)
//{
//	size_t boidCount = arraySize / sizeof(float3);
//
//	int blockCount = boidCount / 256;
//	int threadsInBlockCount;
//
//	boidMoveKernel << <1, 50 >> > (d_boids, boidCount, dt);
//}

__device__ float calculateDistance(float2 startPoint, float2 targetPoint)
{
	float distX = targetPoint.x - startPoint.x;
	distX *= distX;

	float distY = targetPoint.y - startPoint.y;
	distY *= distY;

	return distX + distY;
}

__device__ void updateSeparationFactor(float2 &separationFactor, const float2 &startBoidPosition, const float2 &targetBoidPosition)
{
	separationFactor.x += (startBoidPosition.x - targetBoidPosition.x);
	separationFactor.y += (startBoidPosition.y - targetBoidPosition.y);
}

__device__ void updateAlignmentFactor(float2 &alignmentFactor, const float2 &targetBoidVelocity)
{
	alignmentFactor.x += targetBoidVelocity.x;
	alignmentFactor.y += targetBoidVelocity.y;
}

__device__ void updateCohesionFactor(float2 &cohesionFactor, const float2 &targetBoidPosition)
{
	cohesionFactor.x += targetBoidPosition.x;
	cohesionFactor.y += targetBoidPosition.y;
}

__device__ float2 normalizeVector(const float2 &vector)
{
	float2 result = vector;

	float length = result.x * result.x + result.y * result.y;
	length = sqrtf(length);

	if (isnan(result.x / length) || isnan(result.y / length))
	{
		return { sqrtf(2) / 2.0, sqrtf(2) / 2.0 };
	}

	return	{ result.x / length, result.y / length };
}

__device__ float2 getMovementFromFactors(float2 sumOfFactors, float refreshRateCoefficient)
{
	float2 movement;
	//float angle;

	movement.x = refreshRateCoefficient * sumOfFactors.x;
	movement.y = refreshRateCoefficient * sumOfFactors.y;

	//if (movement.x != 0 || movement.y != 0)
	//	angle = getAngleFromVector(movement);
	//else
	//	angle = 0;

	//return make_float3(movement.x, movement.y, angle);

	return movement;
}

__device__ float2 getBoidPosition(float4 boidData)
{
	return make_float2(boidData.x, boidData.y);
}

__device__ float2 getBoidVelocity(float4 boidData)
{
	return make_float2(boidData.z, boidData.w);
}

__device__ float4 getUpdatedBoidData(float4 oldBoidData, int windowWidth, int windowHeight, float2 movement = { 0,0 })
{
	float4 result;

	result.z = oldBoidData.z + movement.x;
	result.w = oldBoidData.w + movement.y;

	result.x = fmodf(oldBoidData.x + result.z, windowWidth);
	if (result.x < 0)
		result.x += windowWidth;
	result.y = fmodf(oldBoidData.y + result.w, windowHeight);
	if (result.y < 0)
		result.y += windowHeight;

	return result;
}

__device__ int getCellId(float2 position, int gridWidth, int cellSize)
{
	//printf("x: %f, y: %f, cellSize: %d\n", position.x, position.y, cellSize);
	int cellX = position.x / cellSize;
	int cellY = position.y / cellSize;

	return cellY * gridWidth + cellX;
}

__device__ void getNeighbourCells(int cellId, int gridWidth, int gridHeight, int (&neighbourCells)[9], int &neighbourCellsCount)
{
	neighbourCellsCount = 0;

	int gridSize = gridWidth * gridHeight;

	int centerCellId;
	for (int i = 0; i < 3; i++)
	{
		//Center of current row
		if (i == 0)
		{
			centerCellId = cellId;
		}
		else if (i == 1) //north
		{
			centerCellId = cellId - gridWidth;
			if (centerCellId < 0)
				continue;
				//centerCellId += gridSize;
		}
		else if (i == 2) //south
		{
			centerCellId = cellId + gridWidth;
			if (centerCellId >= gridSize)
				continue;
				//centerCellId -= gridSize;
		}

		neighbourCells[neighbourCellsCount++] = centerCellId; //middle

		if (centerCellId % gridWidth != 0) //west
			neighbourCells[neighbourCellsCount++] = centerCellId - 1;
		//else
		//	neighbourCells[neighbourCellsCount++] = centerCellId + gridWidth - 1;

		if ((centerCellId + 1) % gridWidth != 0) //east
			neighbourCells[neighbourCellsCount++] = centerCellId + 1;
		//else
		//	neighbourCells[neighbourCellsCount++] = centerCellId - gridWidth + 1;
	}
}

__global__ void initializeCellsKernel ( float4 *d_boids,
										size_t boidCount,
										int *d_boidId,
										int *d_cellId,
										int gridWidth,
										int cellSize)
{
	int boidIdx = blockDim.x*blockIdx.x + threadIdx.x;
	if (boidIdx >= boidCount)
		return;

	float2 boidPosition = getBoidPosition(d_boids[boidIdx]);

	d_cellId[boidIdx] = getCellId(boidPosition, gridWidth, cellSize);
	d_boidId[boidIdx] = boidIdx;

	//printf("BoidId: %d, CellId: %d\n", d_boidId[boidIdx], d_cellId[boidIdx]);
}

__global__ void updateCellsBeginKernel (size_t boidCount,
										int *d_boidId,
										int *d_cellId,
										int *d_cellBegin,
										int cellCount)
{
	int tId = blockDim.x*blockIdx.x + threadIdx.x;
	if (tId >= boidCount)
		return;

	//printf("BoidId: %d, CellId: %d\n", d_boidId[tId], d_cellId[tId]);

	if (d_cellId[tId] < 0 || d_cellId[tId] > cellCount)
		return;

	if (tId == 0 || d_cellId[tId - 1] < d_cellId[tId])
	{
		d_cellBegin[d_cellId[tId]] = tId;

		//printf("CellId: %d BeginId: %d\n", d_cellId[tId], d_cellBegin[d_cellId[tId]]);
	}
}


__global__ void moveBoidKernel (float4 *d_boids,
								float4 *d_boidsDoubleBuffer,
								size_t boidCount,
								int *d_boidId,
								int *d_cellId,
								int *d_cellIdDoubleBuffer,
								int *d_cellBegin,
								int gridWidth,
								int gridHeight,
								int cellSize,
								int windowWidth,
								int windowHeight,
								float dt,
								float boidSightRangeSquared)
{
	int tId = blockDim.x*blockIdx.x + threadIdx.x;
	if (tId >= boidCount)
		return;

	float refreshRateCoeeficient = dt / 1000;
	int cellId = d_cellId[tId];
	if (cellId < 0)
		return;

	int boidIdx = d_boidId[tId];

	float2 boidPosition = getBoidPosition(d_boids[boidIdx]);
	float2 boidVelocity = getBoidVelocity(d_boids[boidIdx]);

	float2 separationVector;
	float2 alignmentVector;
	float2 cohesionVector;

	int boidsSeen = 0;

	int neighCells[9];
	int neighCellsCount;
	getNeighbourCells(cellId, gridWidth, gridHeight, neighCells, neighCellsCount);

	for (int i = 0; i < neighCellsCount; i++)
	{
		int neighCellId = neighCells[i];
		int cellBegin = d_cellBegin[neighCellId];
		//printf("cellId: %d, cellbegin: %d\n", neighCellId, cellBegin);

		for (int j = cellBegin; j < boidCount; j++)
		{
			if (d_cellId[j] != neighCellId)
				break;

			int targetBoidIdx = d_boidId[j];
			if (boidIdx == targetBoidIdx)
				continue;

			float distance = calculateDistance(boidPosition, getBoidPosition(d_boids[targetBoidIdx]));

			if (distance > boidSightRangeSquared)
				continue;

			updateSeparationFactor(separationVector, boidPosition, getBoidPosition(d_boids[targetBoidIdx]));
			updateAlignmentFactor(alignmentVector, getBoidVelocity(d_boids[targetBoidIdx]));
			updateCohesionFactor(cohesionVector, getBoidPosition(d_boids[targetBoidIdx]));

			boidsSeen++;
		}
	}
	//free(neighCells);
	//for (size_t j = 0; j < boidCount; j++)
	//{
	//	if (boidIdx == j)
	//		continue;

	//	float distance = calculateDistance(boidPosition, getBoidPosition(d_boids[j]));

	//	if (distance > boidSightRangeSquared)
	//		continue;

	//	updateSeparationFactor(separationVector, boidPosition, getBoidPosition(d_boids[j]));
	//	updateAlignmentFactor(alignmentVector, getBoidVelocity(d_boids[j]));
	//	updateCohesionFactor(cohesionVector, getBoidPosition(d_boids[j]));

	//	boidsSeen++;
	//}
	if (boidsSeen == 0)
	{
		d_boidsDoubleBuffer[boidIdx] = getUpdatedBoidData(d_boids[boidIdx], windowWidth, windowHeight);
		d_cellIdDoubleBuffer[tId] = cellId;
		return;
	}

	float2 sumOfFactors = { 0,0 };

	if (fabs(separationVector.x) > 1e-8 && fabs(separationVector.y) > 1e-8)
	{
		separationVector.x = -separationVector.x;
		separationVector.y = -separationVector.x;
		separationVector = normalizeVector(separationVector);

		sumOfFactors.x += separationVector.x;
		sumOfFactors.y += separationVector.y;
	}

	alignmentVector.x = 0.125 * alignmentVector.x / boidsSeen;
	alignmentVector.y = 0.125 * alignmentVector.y / boidsSeen;
	if (fabs(alignmentVector.x) > 1e-8 && fabs(alignmentVector.y) > 1e-8)
	{
		alignmentVector = normalizeVector(alignmentVector);

		sumOfFactors.x += alignmentVector.x;
		sumOfFactors.y += alignmentVector.y;
	}

	cohesionVector.x = 0.001 * (cohesionVector.x / boidsSeen - boidPosition.x);
	cohesionVector.y = 0.001 * (cohesionVector.y / boidsSeen - boidPosition.y);
	if (fabs(cohesionVector.x) > 1e-8 && fabs(cohesionVector.y) > 1e-8)
	{
		cohesionVector = normalizeVector(cohesionVector);

		sumOfFactors.x += cohesionVector.x;
		sumOfFactors.y += cohesionVector.y;
	}

	float2 movement = getMovementFromFactors(sumOfFactors, refreshRateCoeeficient);

	d_boidsDoubleBuffer[boidIdx] = getUpdatedBoidData(d_boids[boidIdx], windowWidth, windowHeight, movement);

	uint newCellId = getCellId(getBoidPosition(d_boidsDoubleBuffer[boidIdx]), gridWidth, cellSize);
	d_cellIdDoubleBuffer[tId] = newCellId;
	//printf("cellId: %d, new: %d\n", cellId, newCellId);
}

void moveBoidKernelExecutor(float4 *&d_boids,
							float4 *&d_boidsDoubleBuffer,
							size_t &arraySize,
							int *&d_boidId,
							int *&d_cellId,
							int *&d_cellIdDoubleBuffer,
							int *&d_cellBegin,
							int gridWidth,
							int gridHeight,
							int cellSize,
							int cellCount,
							int windowWidth,
							int windowHeight,
							float dt,
							float boidSightRangeSquared)
{
	size_t boidCount = arraySize / sizeof(float4);

	// TODO: do this threads number calculations only once
	int blockCount = boidCount / 256;
	if (boidCount % 256 != 0)
	{
		blockCount++;
	}

	moveBoidKernel<<<blockCount, 256>>>(d_boids, d_boidsDoubleBuffer, boidCount, d_boidId, d_cellId, d_cellIdDoubleBuffer, d_cellBegin, gridWidth, gridHeight, cellSize, windowWidth, windowHeight, dt, boidSightRangeSquared);
	hipDeviceSynchronize();

	hipMemcpy(d_cellId, d_cellIdDoubleBuffer, boidCount * sizeof(int), hipMemcpyDeviceToDevice);
	//_sleep(15);
	thrust::sort_by_key(thrust::device_ptr<int>(d_cellId), thrust::device_ptr<int>(d_cellId + boidCount), thrust::device_ptr<int>(d_boidId));
	hipMemset(d_cellBegin, -1, cellCount* sizeof(int));
	updateCellsBeginKernel << <blockCount, 256 >> > (boidCount, d_boidId, d_cellId, d_cellBegin, cellCount);
	hipDeviceSynchronize();

	hipMemcpy(d_boids, d_boidsDoubleBuffer, arraySize, hipMemcpyDeviceToDevice);
	//printf("-------------------------------\n");
}

void initializeCellsKernelExecutor (float4 *&d_boids,
									size_t &boidArraySize,
									int *&d_boidId,
									int *&d_cellId,
									int *&d_cellBegin,
									int gridWidth,
									int cellSize,
									int cellCount)
{
	size_t boidCount = boidArraySize / sizeof(float4);

	// TODO: do this threads number calculations only once
	int blockCount = boidCount / 256;
	if (boidCount % 256 != 0)
	{
		blockCount++;
	}

	//printf("BEFORE:\n");

	initializeCellsKernel << <blockCount, 256 >> > (d_boids, boidCount, d_boidId, d_cellId, gridWidth, cellSize);
	hipDeviceSynchronize();

	thrust::sort_by_key(thrust::device_ptr<int>(d_cellId), thrust::device_ptr<int>(d_cellId + boidCount), thrust::device_ptr<int>(d_boidId));

	//printf("AFTER:\n");

	hipMemset(d_cellBegin, -1, cellCount * sizeof(int));
	updateCellsBeginKernel << <blockCount, 256 >> > (boidCount, d_boidId, d_cellId, d_cellBegin, cellCount);
	hipDeviceSynchronize();
}



//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
